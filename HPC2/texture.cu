#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// Includes CUDA
#include <hip/hip_runtime.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
// Liam helper functions
#include "liam_helpers.h"

const char *imageFilename = "lena_bw.pgm";
const char *maskName = "edge3x3";
const char *sampleName = "naive";
const char *bench = "false";
const char *overhead = "false";

//Define texture
texture<float, 2, hipReadModeElementType> tex;

void runConvolution(int argc, char **argv);

int main(int argc, char **argv)
{

    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "input",
                                     (char **) &imageFilename);
        }
        if (checkCmdLineFlag(argc, (const char **) argv, "mask"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "mask",
                                     (char **) &maskName);
        }
        if (checkCmdLineFlag(argc, (const char **) argv, "bench"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "bench",
                                     (char **) &bench);
        }
        if (checkCmdLineFlag(argc, (const char **) argv, "overhead"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "overhead",
                                     (char **) &overhead);
        }
    }

    if (strcmp(bench,"false")!=0) {
        //Need to make some images to process
        const char *masks[2];
        masks[0] = "bench3x3";
        masks[1] = "bench5x5";
        for (int k=0; k<2; k++) {
          maskName = masks[k];
          for (int i=512; i<6200; i+=512) {
            float* test = (float *)malloc(i*i*sizeof(float));
            time_t t;
            srand((unsigned)time(&t));
            for (int j=0; j<i*i; j++) {\
                test[j] = rand()/(RAND_MAX*1.0f);
            }
            imageFilename = "test.pgm";
            sdkSavePGM(imageFilename, test, i, i);
            runConvolution(argc,argv);
          }
        }
    }
    else {
      printf("%s starting...\n", sampleName);
      runConvolution(argc, argv);
    }
    hipDeviceReset();
    return 0;
}

__global__ void singlePixel(float* out, float* kernel, int width, int kwidth)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    float total = 0;
    for (int v=0; v<kwidth; v++)
    {
        for (int u=0; u<kwidth; u++)
        {
            total += tex2D(tex, i+u, j+v) * kernel[v*kwidth + u];
        }
    }
    out[j*width + i] = gpuclamp(total,0.0f,1.0f);
}

void runConvolution(int argc, char **argv)
{
    if (strcmp(bench,"false")==0) int devID = findCudaDevice(argc, (const char **) argv);

    // Load image from disk
    float *in = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);
    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    sdkLoadPGM(imagePath, &in, &width, &height);
    unsigned int size = width * height * sizeof(float);
    if (strcmp(bench,"false")==0) printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //Create Timer
    StopWatchInterface *timer = NULL;
    checkCudaErrors(hipDeviceSynchronize());
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    //Get Convolution array
    float* kernel;
    int kwidth = kernelWidth(maskName);
    if (kwidth == -1)
    {
        printf("Invalid mask name: %s\n", maskName);
        exit(EXIT_FAILURE);
    }
    kernel = (float *)malloc(sizeof(float)*kwidth*kwidth);
    loadKernel(maskName, kernel);

    //Create Border
    int half = kwidth/2;
    int bwidth = half*2;
    int bigSize = (width+bwidth)*(height+bwidth)*sizeof(float);
    float * temp = (float *)malloc(bigSize);
    for (int y=0; y<height; y++) {
        for (int x=0; x<width; x++)
        {
            temp[(y+half)*(width+bwidth) + (x+half)] = in[y*height+x];
        }
    }
    /* Initialize edges */
    for (int j=0; j<half; j++) {
      for (int i=0; i<width+bwidth; i++) {
        temp[j*(width+bwidth)+i] = 0.0f;
        temp[(width+bwidth-(1+j))*(width+bwidth)+(i+j)] = 0.0f;
        temp[i*(width+bwidth)+j] = 0.0f;
        temp[i*(width+bwidth)+(width+bwidth-(1+j))] = 0.0f;
      }
    }
    free(in);
    in=temp;

    // Allocate memory for result
    float *out = NULL;
    out = (float *)malloc(size);

    // Allocate and initialize device memory for result
    float *dData = NULL;
    checkCudaErrors(hipMalloc((void **) &dData, size));
    checkCudaErrors(hipMemset(dData, 0, size));

    //Allocate and copy kernel
    float *cuKernel;
    checkCudaErrors(hipMalloc((void **) &cuKernel, kwidth*kwidth*sizeof(float)));
    checkCudaErrors(hipMemcpy(cuKernel,
                               kernel,
                               kwidth*kwidth*sizeof(float),
                               hipMemcpyHostToDevice));

    // Allocate and copy image, bind texture
    hipArray *cuArray;
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMallocArray(&cuArray,
                                    &channelDesc,
                                    width+bwidth,
                                    height+bwidth));
    checkCudaErrors(hipMemcpyToArray(cuArray,
                                      0,
                                      0,
                                      in,
                                      bigSize,
                                      hipMemcpyHostToDevice));
    tex.normalized = false;
    tex.filterMode = hipFilterModePoint;
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    // Perform Convolution
    int threadsx = 16;
    int threadsy = 16;
    dim3 threadsPerBlock(threadsx, threadsy);
    int blockx = width/threadsx;
    int blocky = height/threadsy;
    if (blockx%threadsx != 0) blockx++;
    if (blocky%threadsy != 0) blocky++;
    dim3 numBlocks(blockx,blocky);
    if (strcmp(overhead,"false")==0) singlePixel<<<numBlocks,threadsPerBlock>>>(dData, cuKernel, width, kwidth);

    // Copy back results
    checkCudaErrors(hipMemcpy(out,
                               dData,
                               size,
                               hipMemcpyDeviceToHost));

    //Read timer, print stats
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    int flo = 2*width*height*kwidth*kwidth;
    if (strcmp(bench,"false")==0) {
      printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
      printf("Floating Point Operations: %d\n",flo);
      printf("GFLOPS: %f\n", (flo/(sdkGetTimerValue(&timer)/1000))/1000000000);
    }
    else {
      printf("%d,%d,%f,%d,%f\n",width*width,kwidth*kwidth,sdkGetTimerValue(&timer),flo,(flo/(sdkGetTimerValue(&timer)/1000))/1000000000);
    }
    sdkDeleteTimer(&timer);

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
    sdkSavePGM(outputFilename, out, width, height);
    if (strcmp(bench,"false")==0) printf("Wrote '%s'\n", outputFilename);

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipUnbindTexture(tex));
    checkCudaErrors(hipFreeArray(cuArray));
    checkCudaErrors(hipFree(cuKernel));
    free(kernel);
    free(imagePath);
    free(out);
    free(in);
}
