#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// Includes CUDA
#include <hip/hip_runtime.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
// Liam helper functions
#include "liam_helpers.h"

const char *imageFilename = "lena_bw.pgm";
const char *maskName = "edge3x3";
const char *sampleName = "naive";
const char *bench = "false";
const char *overhead = "false";

//Define threads per block
#define THREADS_X 16
#define THREADS_Y 16

void runConvolution(int argc, char **argv);

int main(int argc, char **argv)
{

    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "input",
                                     (char **) &imageFilename);
        }
        if (checkCmdLineFlag(argc, (const char **) argv, "mask"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "mask",
                                     (char **) &maskName);
        }
        if (checkCmdLineFlag(argc, (const char **) argv, "bench"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "bench",
                                     (char **) &bench);
        }
        if (checkCmdLineFlag(argc, (const char **) argv, "overhead"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "overhead",
                                     (char **) &overhead);
        }
    }

    if (strcmp(bench,"false")!=0) {
        //Need to make some images to process
        const char *masks[2];
        masks[0] = "bench3x3";
        masks[1] = "bench5x5";
        for (int k=0; k<2; k++) {
          maskName = masks[k];
          for (int i=512; i<6200; i+=512) {
            float* test = (float *)malloc(i*i*sizeof(float));
            time_t t;
            srand((unsigned)time(&t));
            for (int j=0; j<i*i; j++) {\
                test[j] = rand()/(RAND_MAX*1.0f);
            }
            imageFilename = "test.pgm";
            sdkSavePGM(imageFilename, test, i, i);
            runConvolution(argc,argv);
          }
        }
    }
    else {
      printf("%s starting...\n", sampleName);
      runConvolution(argc, argv);
    }
    hipDeviceReset();
    return 0;
}

__global__ void singlePixel(float* in, float* out, float* kernel, int width, int kwidth)
{
    extern __shared__ float s[];
    //Global coordinates
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    //position of block's 1st element
    int startx = blockIdx.x * blockDim.x;
    int starty = blockIdx.y * blockDim.y;

    //Put stuff into shared memory
    int half = kwidth/2;
    int bwidth = half*2;
    int swidth = blockDim.x+bwidth;
    int iwidth = width+bwidth;
    int endx = half+startx+blockDim.x;
    int endy = half+starty+blockDim.y;
    /* Put in the middle, excluding the borders */
    s[(threadIdx.y+half)*(swidth)+(threadIdx.x+half)] = in[(j+half)*(iwidth)+(i+half)];
    /*Add border sides */
    //Note: The unfortunate complexity of indices here is due to the mapping of
    //      a portion of the image to a local block, which requires some
    //      coordinate shifting.
    //* Top */
    if (threadIdx.y == 0) {
      for (int k=0; k<half; k++) {
        s[k*(swidth)+(threadIdx.x+half)] = in[(starty+k)*(iwidth)+(i+half)];
      }
    }
    //* Bottom */
    else if (threadIdx.y == blockDim.y-1) {
      for (int k=0; k<half; k++) {
        s[(half+blockDim.y+k)*(swidth)+(threadIdx.x+half)] = in[(endy+k)*(iwidth)+(i+half)];
      }
    }
    //* Left */
    if (threadIdx.x == 0) {
      for (int k=0; k<half; k++) {
        s[(threadIdx.y+half)*(swidth)+k] = in[(j+half)*(iwidth)+(startx+k)];
      }
    }
    //* Right */
    else if (threadIdx.x == blockDim.x-1) {
      for (int k=0; k<half; k++) {
        s[(threadIdx.y+half)*(swidth)+(half+blockDim.x+k)] = in[(j+half)*(iwidth)+(endx+k)];
      }
    }
    //Add corners
    if ((threadIdx.x == 1)&&(threadIdx.y == 1)) {
      for (int k=0; k<half; k++) {
        for (int l=0; l<half; l++) {
          s[k*(swidth)+l] = in[(k+starty)*(iwidth)+(startx+l)];
        }
      }
    }
    else if ((threadIdx.x == blockDim.x-2)&&(threadIdx.y == 1)) {
      for (int k=0; k<half; k++) {
        for (int l=0; l<half; l++) {
          s[k*(swidth)+(half+blockDim.x+l)] = in[(k+starty)*(iwidth)+(endx+l)];
        }
      }
    }
    else if ((threadIdx.x == 1)&&(threadIdx.y == blockDim.y-2)) {
      for (int k=0; k<half; k++) {
        for (int l=0; l<half; l++) {
          s[(k+half+blockDim.y)*(swidth)+l] = in[(k+endy)*(iwidth)+(startx+l)];
        }
      }
    }
    else if ((threadIdx.x == blockDim.x-2)&&(threadIdx.y == blockDim.y-2)) {
      for (int k=0; k<half; k++) {
        for (int l=0; l<half; l++) {
          s[(k+half+blockDim.y)*(swidth)+(half+blockDim.x+l)] = in[(k+endy)*(iwidth)+(endx+l)];
        }
      }
    }
    __syncthreads();

    //Do convolution
    float total = 0;
    for (int v=0; v<kwidth; v++)
    {
        for (int u=0; u<kwidth; u++)
        {
            //total += in[(j+v)*(width+kwidth) + (i+u)] * kernel[v*kwidth + u];
            total += s[(threadIdx.y+v)*(swidth) + (threadIdx.x+u)] * kernel[v*kwidth + u];
        }
    }
    //printf("%f\n",s[181]);
    out[j*width + i] = gpuclamp(total,0.0f,1.0f);
}

void runConvolution(int argc, char **argv)
{
    if (strcmp(bench,"false")==0) int devID = findCudaDevice(argc, (const char **) argv);

    // Load image from disk
    float *in = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);
    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    sdkLoadPGM(imagePath, &in, &width, &height);
    unsigned int size = width * height * sizeof(float);
    if (strcmp(bench,"false")==0) printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //Create Timer
    StopWatchInterface *timer = NULL;
    checkCudaErrors(hipDeviceSynchronize());
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    //Get Convolution array
    float* kernel;
    int kwidth = kernelWidth(maskName);
    if (kwidth == -1)
    {
        printf("Invalid mask name: %s\n", maskName);
        exit(EXIT_FAILURE);
    }
    kernel = (float *)malloc(sizeof(float)*kwidth*kwidth);
    loadKernel(maskName, kernel);

    //Create Border
    int half = kwidth/2;
    int bwidth = half*2;
    int bigSize = (width+bwidth)*(height+bwidth)*sizeof(float);
    float *temp = (float *)malloc(bigSize);
    for (int y=0; y<height; y++) {
        for (int x=0; x<width; x++)
        {
            temp[(y+half)*(width+bwidth) + (x+half)] = in[y*height+x];
        }
    }
    /* Initialize edges */
    for (int j=0; j<half; j++) {
      for (int i=0; i<width+bwidth; i++) {
        temp[j*(width+bwidth)+i] = 0.0f;
        temp[(width+bwidth-(1+j))*(width+bwidth)+(i+j)] = 0.0f;
        temp[i*(width+bwidth)+j] = 0.0f;
        temp[i*(width+bwidth)+(width+bwidth-(1+j))] = 0.0f;
      }
    }
    free(in);
    in=temp;

    // Allocate memory for result
    float *out = NULL;
    out = (float *)malloc(size);

    // Allocate and initialize device memory for result
    float *dData = NULL;
    checkCudaErrors(hipMalloc((void **) &dData, size));
    checkCudaErrors(hipMemset(dData, 0, size));

    //Allocate and copy kernel
    float *cuKernel;
    checkCudaErrors(hipMalloc((void **) &cuKernel, kwidth*kwidth*sizeof(float)));
    checkCudaErrors(hipMemcpy(cuKernel,
                               kernel,
                               kwidth*kwidth*sizeof(float),
                               hipMemcpyHostToDevice));

    // Allocate and copy image
    float *cuArray;
    checkCudaErrors(hipMalloc((void **) &cuArray, bigSize));
    checkCudaErrors(hipMemcpy(cuArray,
                               in,
                               bigSize,
                               hipMemcpyHostToDevice));

    // Perform Convolution
    dim3 threadsPerBlock(THREADS_X, THREADS_Y);
    int blockx = width/THREADS_X;
    int blocky = height/THREADS_Y;
    if (blockx%THREADS_X != 0) blockx++;
    if (blocky%THREADS_Y != 0) blocky++;
    dim3 numBlocks(blockx,blocky);
    int sharedmemsize = (THREADS_X+bwidth)*(THREADS_Y+bwidth)*sizeof(float);
    if (strcmp(overhead,"false")==0) singlePixel<<<numBlocks,threadsPerBlock,sharedmemsize>>>(cuArray, dData, cuKernel, width, kwidth);

    // Copy back results
    checkCudaErrors(hipMemcpy(out,
                               dData,
                               size,
                               hipMemcpyDeviceToHost));

    //Read timer, print stats
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    int flo = 2*width*height*kwidth*kwidth;
    if (strcmp(bench,"false")==0) {
      printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
      printf("Floating Point Operations: %d\n",flo);
      printf("GFLOPS: %f\n", (flo/(sdkGetTimerValue(&timer)/1000))/1000000000);
    }
    else {
      printf("%d,%d,%f,%d,%f\n",width*width,kwidth*kwidth,sdkGetTimerValue(&timer),flo,(flo/(sdkGetTimerValue(&timer)/1000))/1000000000);
    }
    sdkDeleteTimer(&timer);

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
    sdkSavePGM(outputFilename, out, width, height);
    if (strcmp(bench,"false")==0) printf("Wrote '%s'\n", outputFilename);

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFree(cuArray));
    checkCudaErrors(hipFree(cuKernel));
    free(kernel);
    free(imagePath);
    free(out);
    free(in);
}
